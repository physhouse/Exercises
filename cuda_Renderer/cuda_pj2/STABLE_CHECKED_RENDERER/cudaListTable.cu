#include <hip/hip_runtime.h>
#include "cudaListTable.h"
#include "exclusiveScan.h"
#include <stdlib.h>
#include <stdio.h>

void initializeTable(ListTable& table, int entries)
{
  table.count = entries;
  int rounded_length = nextPow2(entries);

  hipMalloc((void**)&table.size, rounded_length * sizeof(int));
  hipMemset(table.size, 0, rounded_length * sizeof(int*));
  hipMalloc((void**)&table.index, rounded_length * sizeof(int));
  hipMemset(table.size, 0, rounded_length * sizeof(int*));
}

void generateScan(ListTable& table)  //return the total number of entries
{
  int entries = table.count;
  exclusive_scan(table.size, entries, table.index);

  int sum = 0;
  int last = 0;
  hipMemcpy(&sum, table.index + entries - 1, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&last, table.size + entries - 1, sizeof(int), hipMemcpyDeviceToHost);

  table.elements = sum + last;
  printf("elements: %d, counts: %d\n", table.elements, table.count);
  hipFree(table.entry);
  hipMalloc((void**)&table.entry, sizeof(int) * table.elements);
}

void freeTable(ListTable& table)
{
  hipFree(table.entry);
  hipFree(table.index);
  hipFree(table.size);
}

void copyTable2Host(const ListTable& table, ListTable& hostTable)
{
  hostTable.count = table.count;
  hostTable.elements = table.elements;
  hostTable.size  = (int*) malloc(sizeof(int) * table.count);
  hostTable.index  = (int*) malloc(sizeof(int) * table.count);
  hostTable.entry  = (int*) malloc(sizeof(int*) * table.elements);
  hipMemcpy(hostTable.size, table.size, table.count * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(hostTable.index, table.index, table.count * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(hostTable.entry, table.entry, table.elements * sizeof(int), hipMemcpyDeviceToHost);
}

void deleteHost(ListTable& host)
{
  free(host.entry);
  free(host.index);
  free(host.size);
}

void checkHost(ListTable& host)
{
  for (int i=0; i<host.count; i++)
  {
    int start = host.index[i];
    for (int j=0; j<3; j++)
      printf("%d ", host.entry[start + j]);
    printf(" -> %d\n", i);
  } 
}
