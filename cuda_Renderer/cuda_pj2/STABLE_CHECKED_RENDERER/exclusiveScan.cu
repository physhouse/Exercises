#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "exclusiveScan.h"
#include "stdio.h"

__global__ void excScan(int* device_start, int* device_result, int n)
{
   //Blelloch implementation of exclusive scan
   extern __shared__ int temp[];

   int thid = blockDim.x * blockIdx.x + threadIdx.x;
   int tid  = threadIdx.x;

   temp[2*tid] = device_start[2*thid];
   temp[2*tid + 1] = device_start[2*thid + 1];

   //Up Sweeping
   int offset = 1;
   for (int d = n>>1; d > 0; d >>= 1)
   {
     __syncthreads();
     if (tid < d)
     {
       int a = offset * (2*tid + 1) - 1;
       int b = offset * (2*tid + 2) - 1;
       temp[b] += temp[a];
     }
     offset *= 2;
   }

   if (tid == 0) temp[n-1] = 0;

   //Down Sweeping Algorithms
   offset = n;
   for (int d = 1; d < n; d *= 2)
   {
      __syncthreads();
      if (tid < d)
      {
        //int a = offset * (2*tid + 1) - 1;
        //int b = offset * (2*tid + 2) - 1;

        int a = n - tid * offset - 1;
        int b = n - tid * offset - 1 - offset / 2;

        int tmp = temp[a];
        temp[a] += temp[b];
        temp[b] = tmp;
      }
      offset >>= 1;
   }
   __syncthreads();

   device_result[2*thid] = temp[2*tid];
   device_result[2*thid + 1] = temp[2*tid + 1];
}

__global__ void postSum(int* device_result, int* scanBlocks, int length)
{
   int thid = blockDim.x * blockIdx.x + threadIdx.x;
   int bid = blockIdx.x;

   device_result[2*thid] += scanBlocks[bid];
   device_result[2*thid + 1] += scanBlocks[bid];
}

__global__ void preSum(int* device_start, int* device_result, int* sumBlocks, int n)
{
   int tid = threadIdx.x;
   int bid = blockIdx.x; // Recursive
   //if (tid == 0) sumBlocks[tid] = device_start[n*(tid+1) - 1] + device_result[n*(tid+1) - 1];
   if (tid == 0) sumBlocks[bid] = device_start[n*(bid+1) - 1] + device_result[n*(bid+1) - 1];
}

void exclusive_scan(int* device_start, int length, int* device_result)
{
    /* Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the input and output in device memory,
     * but this is host code -- you will need to declare one or more CUDA 
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the input and the output arrays are sized to accommodate the next
     * power of 2 larger than the input.
     */
    int blocksize = 512; //512;
    while (length < 2 * blocksize)
    {
       blocksize /= 2;
    }
    //length = nextPow2(length);
    int nblocks = (length/2 + blocksize - 1) / blocksize;
    int bytesShared = 0;

    int* sumBlocks;
    int* scanBlocks;
    int sizeNext = nextPow2(nblocks);
    hipMalloc((void**)&sumBlocks, sizeof(int) * sizeNext);
    hipMalloc((void**)&scanBlocks, sizeof(int) * sizeNext);
    hipMemset(sumBlocks, 0, sizeof(int) * nblocks);
    hipMemset(scanBlocks, 0, sizeof(int) * nblocks);

    bytesShared = sizeof(int) * 2 * blocksize;
    excScan<<<nblocks, blocksize, bytesShared>>>(device_start, device_result, 2*blocksize);

    if (nblocks > 1)
    {
      preSum<<<nblocks, blocksize>>>(device_start, device_result, sumBlocks, 2*blocksize);

      exclusive_scan(sumBlocks, nblocks, scanBlocks);

      postSum<<<nblocks, blocksize>>>(device_result, scanBlocks, 2*blocksize);
    }

    hipFree(sumBlocks);
    hipFree(scanBlocks);
}

void cudaScan(int* inarray, int length, int* resultarray)
{
  int* device_result;
  int* device_input;
  int rounded = nextPow2(length);
  hipMalloc((void**)&device_result, sizeof(int) * rounded);
  hipMalloc((void**)&device_input, sizeof(int) * rounded);
  hipMemcpy(device_input, inarray, sizeof(int) * length, hipMemcpyHostToDevice);

  hipMemcpy(device_result, inarray, sizeof(int) * length, hipMemcpyHostToDevice); 

  exclusive_scan(device_input, length, device_result);
  
  hipMemcpy(resultarray, device_result, length * sizeof(int), hipMemcpyDeviceToHost);
}
